#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGenerator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/DistributionTemplates.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>

#include <ATen/native/Distributions.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/LegacyTHFunctionsCUDA.h>

#include <THC/THCGeneral.h>
#include <THC/THCApply.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cstdint>
#include <cmath>
#include <limits>
#include <utility>
#include <type_traits>


namespace {

template <typename scalar_t>
void poisson_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& lambda,
    std::pair<uint64_t, uint64_t> seeds) {
  at::TensorIterator iter;
  iter.add_output(ret);
  iter.add_input(lambda);
  iter.build();
  bool initialized = false;
  hiprandStatePhilox4_32_10_t state;
  at::native::gpu_kernel(iter,
    [seeds, state, initialized] GPU_LAMBDA (scalar_t lambda) mutable -> scalar_t {
      #if defined(__CUDA_ARCH__) || defined(__HIP_PLATFORM_HCC__)
      if (!initialized) {
        hiprand_init(
            seeds.first,
            blockIdx.x * blockDim.x + threadIdx.x,
            seeds.second,
            &state);
        initialized = true;
      }
      return static_cast<scalar_t>(hiprand_poisson(&state, lambda));
      #else
      return static_cast<scalar_t>(std::nan(""));  // just to avoid compiler warning
      #endif
    });
}

struct curand_uniform_wrapper {
  hiprandStatePhilox4_32_10_t &state;
  __device__ curand_uniform_wrapper(hiprandStatePhilox4_32_10_t &state): state(state) {}
  __device__ float operator()() {
    return hiprand_uniform(&state);
  }
};

struct curand_normal_wrapper {
  hiprandStatePhilox4_32_10_t &state;
  __device__ curand_normal_wrapper(hiprandStatePhilox4_32_10_t &state): state(state) {}
  __device__ float operator()() {
    return hiprand_normal(&state);
  }
};

template <typename scalar_t>
void gamma_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& alpha,
    std::pair<uint64_t, uint64_t> seeds) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  at::TensorIterator iter;
  iter.add_output(ret);
  iter.add_input(alpha);
  iter.build();

  at::native::gpu_kernel(iter,
    [seeds] GPU_LAMBDA (scalar_t alpha) {
      #if defined(__CUDA_ARCH__) || defined(__HIP_PLATFORM_HCC__)
      hiprandStatePhilox4_32_10_t state;
      hiprand_init(
          seeds.first,
          blockIdx.x * blockDim.x + threadIdx.x,
          seeds.second,
          &state);

      auto uniform_lambda = curand_uniform_wrapper(state);
      BaseSampler<accscalar_t, decltype(uniform_lambda)> standard_uniform(uniform_lambda);

      auto normal_lambda = curand_normal_wrapper(state);
      BaseSampler<accscalar_t, decltype(normal_lambda)> standard_normal(normal_lambda);
      auto sample = sample_gamma<scalar_t, accscalar_t, decltype(uniform_lambda), decltype(normal_lambda)>(alpha, standard_uniform, standard_normal);
      auto min_value = std::numeric_limits<scalar_t>::min();
      return (min_value > sample) ? min_value : sample;
      #else
      return alpha;  //useless
      #endif
    });
}

template<typename scalar_t>
void dirichlet_scalar_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& gamma) {
  auto gamma_sum = gamma.sum(-1, true);
  at::TensorIterator iter;
  iter.add_output(ret);
  iter.add_input(gamma);
  iter.add_input(gamma_sum);
  iter.build();
  at::native::gpu_kernel(iter,
    [] GPU_LAMBDA (scalar_t gamma, scalar_t gamma_sum) {
      auto ret_val = gamma / gamma_sum;
      auto min_value = std::numeric_limits<scalar_t>::min();
      auto max_value = 1 - std::numeric_limits<scalar_t>::epsilon();
      ret_val = (min_value > ret_val) ? min_value : ret_val;
      ret_val = (max_value < ret_val) ? max_value : ret_val;
      return ret_val;
    });
}

} // namespace

namespace at { namespace native {

Tensor _s_poisson_cuda(const Tensor& lambda, Generator gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(20);
  }
  Tensor ret = at::empty(lambda.sizes(), lambda.options());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "poisson_cuda", [&] {
    poisson_cuda_kernel<scalar_t>(ret, lambda, rng_engine_inputs);
  });
  return ret;
}

Tensor _s_gamma_cuda(const Tensor& alpha, Generator gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(10);
  }
  Tensor ret = at::empty(alpha.sizes(), alpha.options());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "gamma_cuda", [&] {
     gamma_cuda_kernel<scalar_t>(ret, alpha, rng_engine_inputs);
   });
  return ret;
}

Tensor _s_dirichlet_cuda(const Tensor& alpha, Generator gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(10);
  }
  Tensor ret = at::empty(alpha.sizes(), alpha.options());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "dirichlet", [&] {
    Tensor gamma = at::empty(alpha.sizes(), alpha.options());
    gamma_cuda_kernel<scalar_t>(gamma, alpha, rng_engine_inputs);
    dirichlet_scalar_cuda_kernel<scalar_t>(ret, gamma);
  });
  return ret;
}

Tensor _standard_gamma_grad_cuda(const Tensor& self, const Tensor& output) {
  Tensor ret = at::empty(self.sizes(), self.options());
  TensorIterator iter;
  iter.add_output(ret);
  iter.add_input(self);
  iter.add_input(output);
  iter.build();
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "_standard_gamma_grad_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    gpu_kernel(iter,
      [] GPU_LAMBDA (scalar_t self_val, scalar_t output_val) {
        return standard_gamma_grad_one<scalar_t, accscalar_t>(self_val, output_val);
      });
  });
  return ret;
}

Tensor _dirichlet_grad_cuda(const Tensor& x, const Tensor& alpha, const Tensor& total) {
  Tensor ret = at::empty(x.sizes(), x.options());
  TensorIterator iter;
  iter.add_output(ret);
  iter.add_input(x);
  iter.add_input(alpha);
  iter.add_input(total);
  iter.build();
  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "_dirichlet_grad_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    gpu_kernel(iter,
      [] GPU_LAMBDA (scalar_t x_val, scalar_t alpha_val, scalar_t total_val) -> scalar_t {
        return dirichlet_grad_one<scalar_t, accscalar_t>(x_val, alpha_val, total_val);
      });
  });
  return ret;
}

}} // namespace at::native
